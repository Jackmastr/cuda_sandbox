#include "hip/hip_runtime.h"
#include <stdio.h>

__global__
void saxpy(int n, float a, float *x, float *y)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = index; i < n; i += stride)
	{
		y[i] = a*x[i] + y[i];
	}
}


int main(void)
{
	int N = 1<<20;
	float *x, *y, *d_x, *d_y;
	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));

	hipMalloc(&d_x, N*sizeof(float));
	hipMalloc(&d_y, N*sizeof(float));

	for (int i = 0; i < N; i++)
	{
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipMemcpyKind hTD = hipMemcpyHostToDevice;

	hipMemcpy(d_x, x, N*sizeof(float), hTD);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

	// Perform SAXPY on 1 Million Elements
	hipEventRecord(start);
	saxpy<<<(N+255)/256, 256>>>(N, 2.0, d_x, d_y);
	hipEventRecord(stop);

	// Error Checking
	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));

	hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("time to complete: %f ms.\n", milliseconds);

	// Testing
	float maxError = 0.0f;
	for (int i = 0; i < N; i++)
	{
		maxError = max(maxError, abs(y[i]-4.0f));
	}
	printf("Max error: %f\n", maxError);
	printf("Effective Bandwidth (GB/s): %f\n", (N*4*3)/(milliseconds*1e6));

	hipFree(d_x);
	hipFree(d_y);
	free(x);
	free(y);
}
